#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel function for SAXPY: Single-Precision A*X Plus Y.
__global__ void saxpy_parallel(int n, float alpha, float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = alpha * x[i] + y[i]; // Perform the computation
    }

    // Print block number - only print once per block using the thread with threadIdx.x == 0
    if (threadIdx.x == 0) {
        printf("Block number %d\n", blockIdx.x);
        printf("blockDim.x = %d\n",blockDim.x);
    }
    //printf("threadIdx.x = %d\n",threadIdx.x);
}

int main()
{
    const int n = 2000; // Example array size
    float *x, *y; // Pointers for host arrays
    float *d_x, *d_y; // Pointers for device arrays

    // Allocate host memory
    x = (float*)malloc(n * sizeof(float));
    y = (float*)malloc(n * sizeof(float));

    // Initialize host arrays
    for(int i = 0; i < n; i++) {
        x[i] = i;
        y[i] = n - i;
    }

    // Allocate device memory
    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_y, n * sizeof(float));

    // Copy host data to device
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    // Invoke the SAXPY kernel (256 threads per block)
    int nblocks = (n + 255) / 256;
    saxpy_parallel<<<nblocks, 256>>>(n, 2.0f, d_x, d_y);

    //blockDim.x = 256;
    //for (int block = 0; block < nblocks; nblocks++) {
        //blockIdx.x = block;
        //for (int thread = 0; thread < 256; thread++) {
            //threadIdx.x = thread;
            //saxpy_parallel(n, 2.0f, d_x, d_y);
        //}
    //}

    // Copy result back to host
    hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_x);
    hipFree(d_y);

    // Print the content of the array
    for (int i = 0; i < n; ++i) {
        printf("y[%d] = %f\n", i, y[i]);
    }
    // Free host memory
    free(x);
    free(y);

    return 0;
}